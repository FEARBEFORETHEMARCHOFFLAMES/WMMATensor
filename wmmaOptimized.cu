#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_helper.cuh"
#include <cstdlib>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "mma.h";
#include <fstream>

constexpr int TILE_SIZE = 16;

// https://developer.nvidia.com/blog/programming-tensor-cores-cuda-9/#programmatic_access_to_tensor_cores_in_cuda_90
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/#warp-matrix-functions
// https://blog.demofox.org/2017/11/21/floating-point-precision/
// https://blogs.nvidia.com/blog/double-precision-tensor-cores/ -> in hpc alpha?
template<typename T> __global__ void matmuladd_simple(T const* const a, T const* const b, T* const c, 
    const int N, const int K, const int M) {
    for (int row = threadIdx.y + blockIdx.y * blockDim.y;
        row < N;
        row += blockDim.y * gridDim.y) {

        for (int col = threadIdx.x + blockIdx.x * blockDim.x;
            col < M;
            col += blockDim.x * gridDim.x) {
            T result = 0;

            for (int k = 0; k < K; k++) {
                result += a[row * N + k] * b[k * M + col];
            }
            c[row * M + col] += result;
        }
    }
}

//Perform tiled matrix matrix multiplication
//WMMA works on tile sizes of 16
//We divide the matrix into equal sized tiles of 16
//An output tile in the matrix c is the result of C_ij = Sum over k(A_ik * B_kj)
// => basically, it works just like the element wise multiplication. Except that every element is replaced with a 16x16 tile!
template<typename T> __global__ void wmma_kernel(T* a, T* b, T* c, const int N, const int K, const int M) {
    fragment<matrix_a, TILE_SIZE, TILE_SIZE, TILE_SIZE, T, row_major> a_frag;
    fragment<matrix_b, TILE_SIZE, TILE_SIZE, TILE_SIZE, T, row_major> b_frag;
    fragment<accumulator, TILE_SIZE, TILE_SIZE, TILE_SIZE, T> c_frag;

    for (int row = 0; row < M; row += TILE_SIZE) {
        for (int col = 0; col < N; col += TILE_SIZE) {
            fill_fragment(c_frag, 0.0f);

            for (int k = 0; k < K; k += TILE_SIZE) {
                // The last argument is the stride between consecutive rows -> i.e. we load 16 elements from the first row, how many elements to skip to get to the next row?
                // -> amount of columns, K for a and N for b
                // We have to start the load at a 256bit aligned position (16x16=256), do pointer arithmetic to figure out start of tile
                load_matrix_sync(a_frag, &a[row * K + k], K);
                load_matrix_sync(b_frag, &b[k * N + col], N);

                // Continuously sum up tiles
                mma_sync(c_frag, a_frag, b_frag, c_frag);
            }
            // store result of tile sum(c_frag) in the corresponding c tile. N is again the number of columns(stride between rows of the tile)
            store_matrix_sync(&c[row * N + col], c_frag, N, mem_row_major);
        }
    }
}

template<typename T> void runMatmulSimple(T* h_a, T* h_b, T* h_c, T* result, const int N, const int K, const int M, const int threadsPerSM, const int blocks) {
    T* h_d = new T[N * M];
    T* d_a, * d_b, * d_c;
    CHECK_CUDA(hipMalloc(&d_a, N * K * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_b, K * M * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_c, N * M * sizeof(T)));

    CHECK_CUDA(hipMemcpy(d_a, h_a, N * K * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, h_b, K * M * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_c, h_c, N * M * sizeof(T), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(threadsPerSM);
    dim3 blocksPerGrid(blocks);

    matmuladd_simple<T><<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N, K, M);

    CHECK_CUDA(hipMemcpy(result, d_c, N * M * sizeof(T), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipDeviceSynchronize());
}

template<typename T> void runWMMA(T* h_a, T* h_b, T* h_c, T* result, const int N, const int K, const int M, const int threadsPerSM, const int blocks) {
    T* h_d = new T[N * M];
    T* d_a, * d_b, * d_c;
    CHECK_CUDA(hipMalloc(&d_a, N * K * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_b, K * M * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_c, N * M * sizeof(T)));

    CHECK_CUDA(hipMemcpy(d_a, h_a, N * K * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, h_b, K * M * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_c, h_c, N * M * sizeof(T), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(threadsPerSM);
    dim3 blocksPerGrid(blocks);

    wmma_kernel<T><<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N, K, M);

    CHECK_CUDA(hipMemcpy(result, d_c, N * M * sizeof(T), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipDeviceSynchronize());
}

int main()
{
    const int dev = 0;
    int threadsPerSM = 128;
    int blocks = 1;
    std::cout << getCUDADeviceInformations(dev).str() << "\n\n";
    std::srand(1337);

    int sizes[5] = { 256, 512, 1024, 2048, 4096 };
    int repetitions = 10;
    int N, K, M = 0;
    std::ofstream myfile;
    half divisor_half = (half)(RAND_MAX + 1u);
    double divisor_double = (double)(RAND_MAX + 1u);
    myfile.open("DoubleSimple_vs_HalfWMMA_1_2.csv");
    for (int s = 0; s < 5; s++) {
        N = sizes[s];
        M = sizes[s];
        K = sizes[s];
        threadsPerSM = std::min(256, sizes[s] / 2);
        blocks = sizes[s] / threadsPerSM / 2;
        half* h_a = new half[N * K]; // N hoch, K breit
        double* h_a_2 = new double[N * K]; // N hoch, K breit
        half* h_b = new half[K * M]; // K hoch, M breit
        double* h_b_2 = new double[K * M]; // K hoch, M breit
        half* h_c = new half[N * M]; // N hoch, M breit
        double* h_c_2 = new double[N * M]; // N hoch, M breit
        for (int i = 0; i < repetitions; i++) {
            for (size_t i = 0; i < N * K; i++) {
                int randomNumber = std::rand();
                h_a[i] = (half)1.0f + (half)randomNumber / divisor_half;
                h_a_2[i] = 1.0f + (double)randomNumber / divisor_double;
            }
            for (size_t i = 0; i < K * M; i++) {
                int randomNumber = std::rand();
                h_b[i] = (half)1.0f + (half)randomNumber / divisor_half;
                h_b_2[i] = 1.0f + (double)randomNumber / divisor_double;
            }
            for (size_t i = 0; i < N * M; i++) {
                //It is not possible to load things directly into c in wmma!
                h_c[i] = 0.0f;
                h_c_2[i] = 0.0f;
            }
            std::cout << "Done with generating data" << std::endl;
            half* result1 = new half[N * M];
            runWMMA<half>(h_a, h_b, h_c, result1, N, K, M, threadsPerSM, blocks);
            std::cout << "Done with half simple" << std::endl;
            double* result2 = new double[N * M];
            runMatmulSimple<double>(h_a_2, h_b_2, h_c_2, result2, N, K, M, threadsPerSM, blocks);
            std::cout << "Done with float simple"<< std::endl;
            
            long double totalDiff = totalDifference(result2, result1, N, M);
            std::cout << "Calculation done for size " << sizes[s] << " and repetition " << i << std::endl;
            std::cout << "Total difference between regular and WMMA result: " << totalDiff << std::endl;
            std::cout << "Average difference between regular and WMMA result: " << totalDiff/ (long double)(N * M) << std::endl;
            std::cout << "The range of input numbers was half numbers in [0,6] " << std::endl << std::endl;
            myfile << totalDiff << ";";
        }
        myfile << "\r\n";
    }
    myfile.close();
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
