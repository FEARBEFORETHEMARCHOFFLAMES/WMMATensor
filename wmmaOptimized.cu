#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_helper.cuh"
#include <cstdlib>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "mma.h";
#include <fstream>

constexpr int TILE_SIZE = 16;

// WMMA guide/article: https://developer.nvidia.com/blog/programming-tensor-cores-cuda-9/#programmatic_access_to_tensor_cores_in_cuda_90
// Nvidia programming guide: https://docs.nvidia.com/cuda/cuda-c-programming-guide/#warp-matrix-functions
// Float precision: https://blog.demofox.org/2017/11/21/floating-point-precision/
// Double precision in tensors: https://blogs.nvidia.com/blog/double-precision-tensor-cores/ -> in hpc alpha?
// Which fragment sizes are supported?: https://forums.developer.nvidia.com/t/why-does-wmma-and-mma-support-different-matrix-tile-size/271067 -> https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#warp-level-matrix-instructions-wmma-mma
template<typename T> __global__ void matmuladd_simple(T const* const a, T const* const b, T* const c, 
    const int N, const int K, const int M) {
    for (int row = threadIdx.y + blockIdx.y * blockDim.y;
        row < N;
        row += blockDim.y * gridDim.y) {

        for (int col = threadIdx.x + blockIdx.x * blockDim.x;
            col < M;
            col += blockDim.x * gridDim.x) {
            T result = 0;

            for (int k = 0; k < K; k++) {
                result += a[row * K + k] * b[k * M + col];
            }
            c[row * M + col] += result;
        }
    }
}

using namespace nvcuda::wmma;
//Perform tiled matrix matrix multiplication
//WMMA works on tile sizes of 16
//We divide the matrix into equal sized tiles of 16
//An output tile in the matrix c is the result of C_ij = Sum over k(A_ik * B_kj)
// => basically, it works just like the element wise multiplication. Except that every element is replaced with a 16x16 tile!#
// If the input matrices do not fit the tiling neatly, we will have to manually calculate the remainder matrix parts
// -> this is probably
template<typename T> __global__ void wmma_kernel(T* a, T* b, T* c, T* a_temp, T* b_temp, const int N, const int K, const int M) {
    fragment<matrix_a, TILE_SIZE, TILE_SIZE, TILE_SIZE, T, row_major> a_frag;
    fragment<matrix_b, TILE_SIZE, TILE_SIZE, TILE_SIZE, T, row_major> b_frag;
    fragment<accumulator, TILE_SIZE, TILE_SIZE, TILE_SIZE, T> c_frag;
    bool printedOnce = false;
    for (int row = 0; row < N; row += TILE_SIZE) {
        for (int col = 0; col < M; col += TILE_SIZE) {
            fill_fragment(c_frag, 0.0f);

            for (int k = 0; k < K; k += TILE_SIZE) {
                if (TILE_SIZE > N - row  // tile size is greater than remaining rows in A
                 || TILE_SIZE > K - k    // tile size is greater than remaining cols in A / remaining rows in B
                 || TILE_SIZE > M - col) { // tile size is greater than remaining cols in B
                //    /*for (int rowLeftover = row + threadIdx.y + blockIdx.y * blockDim.y;
                //        rowLeftover < N;
                //        rowLeftover += blockDim.y * gridDim.y) {

                //        for (int colLeftover = col + threadIdx.x + blockIdx.x * blockDim.x;
                //            colLeftover < M;
                //            colLeftover += blockDim.x * gridDim.x) {
                //            T result = 0;

                //            for (int kLeftover = k; kLeftover < K; kLeftover++) {
                //                result += a[rowLeftover * K + kLeftover] * b[kLeftover * M + colLeftover];
                //            }
                //            c[rowLeftover * M + colLeftover] += result;
                //        }
                //    }*/
                    auto lane_id = threadIdx.x % 32;
                    // we want to populate the a_temp and b_temp vector with data from a/b AND padded zeros
                    for (int i = 0; i < 8; i++) {
                        int targetRow = row + lane_id / 16 + 2 * i;
                        int targetColumn = k + lane_id % 16;
                        if (targetRow < N && targetColumn < K) {
                            a_temp[(lane_id/16 + 2 * i) * 16 + lane_id % 16] = a[targetRow * K + targetColumn];
                        }
                        else {
                            a_temp[(lane_id / 16 + 2 * i) * 16 + lane_id % 16] = 0;
                        }

                        targetRow = k + lane_id / 16 + 2 * i;
                        targetColumn = col + lane_id % 16;
                        if (targetRow < K && targetColumn < M) {
                            b_temp[i * 32 + lane_id] = b[targetRow * M + targetColumn];
                        }
                        else {
                            b_temp[i * 32 + lane_id] = 0;
                        }
                    }


                    if (threadIdx.x == 0 && blockIdx.x == 0 && !printedOnce) {
                        printf("a:\r\n[");
                        for (int y = 0; y < 16; y++) {
                            for (int x = 0; x < 16; x++) {
                                printf("%f ", __half2float(a_temp[y * 16 + x]));
                            }
                            if (y == 15) {
                                printf("]\r\n");
                            }
                            else {
                                printf("\r\n ");
                            }
                        }
                        printf("b:\r\n[");
                        for (int y = 0; y < 16; y++) {
                            for (int x = 0; x < 16; x++) {
                                printf("%f ", __half2float(b_temp[y * 16 + x]));
                            }
                            if (y == 15) {
                                printf("]\r\n");
                            }
                            else {
                                printf("\r\n ");
                            }
                        }
                        printedOnce = true;
                    }
                    __syncwarp();
                    load_matrix_sync(a_frag, a_temp, 0);
                    load_matrix_sync(b_frag, b_temp, 0);
                    mma_sync(c_frag, a_frag, b_frag, c_frag);
                }
                else {
                    // The last argument is the stride between consecutive rows -> i.e. we load 16 elements from the first row, how many elements to skip to get to the next row?
                    // -> amount of columns, K for a and N for b
                    // We have to start the load at a 256bit aligned position (16x16=256), do pointer arithmetic to figure out start of tile
                    load_matrix_sync(a_frag, &a[row * K + k], K);
                    load_matrix_sync(b_frag, &b[k * M + col], M);
                    mma_sync(c_frag, a_frag, b_frag, c_frag);

                    // Continuously sum up tiles
                }
            }
            // store result of tile sum(c_frag) in the corresponding c tile. N is again the number of columns(stride between rows of the tile)
            store_matrix_sync(&c[row * M + col], c_frag, M, mem_row_major);
        }
    }
}

template<typename T> void runMatmulSimple(T* h_a, T* h_b, T* h_c, T* result, const int N, const int K, const int M, const int threadsPerSM, const int blocks) {
    T* h_d = new T[N * M];
    T* d_a, * d_b, * d_c;
    CHECK_CUDA(hipMalloc(&d_a, N * K * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_b, K * M * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_c, N * M * sizeof(T)));

    CHECK_CUDA(hipMemcpy(d_a, h_a, N * K * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, h_b, K * M * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_c, h_c, N * M * sizeof(T), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(threadsPerSM);
    dim3 blocksPerGrid(blocks);

    matmuladd_simple<T><<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N, K, M);

    CHECK_CUDA(hipMemcpy(result, d_c, N * M * sizeof(T), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipDeviceSynchronize());
}

template<typename T> void runWMMA(T* h_a, T* h_b, T* h_c, T* result, const int N, const int K, const int M, const int threadsPerSM, const int blocks) {
    T* h_d = new T[N * M];
    T* d_a, * d_b, * d_c;
    T* d_a_temp, *d_b_temp;
    CHECK_CUDA(hipMalloc(&d_a, N * K * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_b, K * M * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_c, N * M * sizeof(T)));

    CHECK_CUDA(hipMalloc(&d_a_temp, TILE_SIZE * TILE_SIZE * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_b_temp, TILE_SIZE * TILE_SIZE * sizeof(T)));

    CHECK_CUDA(hipMemcpy(d_a, h_a, N * K * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, h_b, K * M * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_c, h_c, N * M * sizeof(T), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(threadsPerSM);
    dim3 blocksPerGrid(blocks);

    wmma_kernel<T><<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, d_a_temp, d_b_temp, N, K, M);

    CHECK_CUDA(hipMemcpy(result, d_c, N * M * sizeof(T), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipDeviceSynchronize());
}

void createTestFile() {
    const int dev = 0;
    int threadsPerSM = 128;
    int blocks = 1;
    std::cout << getCUDADeviceInformations(dev).str() << "\n\n";
    std::srand(1337);

    int sizes[5] = { 256, 512, 1024, 2048, 4096 };
    int repetitions = 10;
    int N, K, M = 0;
    std::ofstream myfile;
    half divisor_half = (half)(RAND_MAX + 1u);
    double divisor_double = (double)(RAND_MAX + 1u);
    myfile.open("DoubleSimple_vs_HalfWMMA_1_2.csv");
    for (int s = 0; s < 5; s++) {
        N = sizes[s];
        M = sizes[s];
        K = sizes[s];
        threadsPerSM = std::min(256, sizes[s] / 2);
        blocks = sizes[s] / threadsPerSM / 2;
        half* h_a = new half[N * K]; // N hoch, K breit
        double* h_a_2 = new double[N * K]; // N hoch, K breit
        half* h_b = new half[K * M]; // K hoch, M breit
        double* h_b_2 = new double[K * M]; // K hoch, M breit
        half* h_c = new half[N * M]; // N hoch, M breit
        double* h_c_2 = new double[N * M]; // N hoch, M breit
        for (int i = 0; i < repetitions; i++) {
            for (size_t i = 0; i < N * K; i++) {
                int randomNumber = std::rand();
                h_a[i] = (half)1.0f + (half)randomNumber / divisor_half;
                h_a_2[i] = 1.0f + (double)randomNumber / divisor_double;
            }
            for (size_t i = 0; i < K * M; i++) {
                int randomNumber = std::rand();
                h_b[i] = (half)1.0f + (half)randomNumber / divisor_half;
                h_b_2[i] = 1.0f + (double)randomNumber / divisor_double;
            }
            for (size_t i = 0; i < N * M; i++) {
                //It is not possible to load things directly into c in wmma!
                h_c[i] = 0.0f;
                h_c_2[i] = 0.0f;
            }
            std::cout << "Done with generating data" << std::endl;
            half* result1 = new half[N * M];
            runWMMA<half>(h_a, h_b, h_c, result1, N, K, M, threadsPerSM, blocks);
            std::cout << "Done with half simple" << std::endl;
            double* result2 = new double[N * M];
            runMatmulSimple<double>(h_a_2, h_b_2, h_c_2, result2, N, K, M, threadsPerSM, blocks);
            std::cout << "Done with float simple" << std::endl;

            long double totalDiff = totalDifference(result2, result1, N, M);
            std::cout << "Calculation done for size " << sizes[s] << " and repetition " << i << std::endl;
            std::cout << "Total difference between regular and WMMA result: " << totalDiff << std::endl;
            std::cout << "Average difference between regular and WMMA result: " << totalDiff / (long double)(N * M) << std::endl;
            std::cout << "The range of input numbers was half numbers in [0,6] " << std::endl << std::endl;
            myfile << totalDiff << ";";
        }
        myfile << "\r\n";
    }
    myfile.close();
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }
}

int main()
{
    const int dev = 0;
    std::cout << getCUDADeviceInformations(dev).str() << "\n\n";
    int M = 1025, K = 1025, N = 1025;
    int threadsPerSM = 256;
    int blocks = 2;
    
    half* h_a = new half[N * K]; // N hoch, K breit
    float* h_a_2 = new float[N * K]; // N hoch, K breit
    half* h_b = new half[K * M]; // K hoch, M breit
    float* h_b_2 = new float[K * M]; // K hoch, M breit
    half* h_c = new half[N * M]; // N hoch, M breit
    float* h_c_2 = new float[N * M]; // N hoch, M breit

    half divisor_half = (half)(RAND_MAX + 1u);
    float divisor_double = (float)(RAND_MAX + 1u);

    for (size_t i = 0; i < N * K; i++) {
        int randomNumber = std::rand();
        h_a[i] = (half)randomNumber / divisor_half;
        h_a_2[i] = (float)randomNumber / divisor_double;
    }

    for (size_t i = 0; i < K * M; i++) {
        int randomNumber = std::rand();
        h_b[i] = (half)randomNumber / divisor_half;
        h_b_2[i] = (float)randomNumber / divisor_double;
    }

    std::cout << "h_a: [ ";
    for (int i = 0; i < N; i++) {
        std::cout << __half2float(h_a[i * N + K - 1]) << " ";
    }
    std::cout << "]\r\n";

    std::cout << "h_b: [ ";
    for (int i = 0; i < N; i++) {
        std::cout << __half2float(h_b[(K-1) * M + i]) << " ";
    }
    std::cout << "]\r\n";
    for (size_t i = 0; i < N * M; i++) {
        //It is not possible to load things directly into c in wmma!
        h_c[i] = 0.0f;
        h_c_2[i] = 0.0f;
    }
    half* result1 = new half[N * M];
    runWMMA<half>(h_a, h_b, h_c, result1, N, K, M, threadsPerSM, blocks);
    float* result2 = new float[N * M];
    runMatmulSimple<float>(h_a_2, h_b_2, h_c_2, result2, N, K, M, threadsPerSM, blocks);

    long double totalDiff = totalDifference(result2, result1, N, M);
    std::cout << "Total difference between simple cuda float and WMMA result: " << totalDiff << std::endl;
    std::cout << "Average difference between regular and WMMA result: " << totalDiff / (long double)(N * M) << std::endl;
    return 0;
}
