#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_helper.cuh"
#include <cstdlib>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "mma.h";

constexpr auto N = 4;
constexpr auto K = 4;
constexpr auto M = 4;

constexpr auto threadsPerSM = 128;

// https://docs.nvidia.com/cuda/cuda-c-programming-guide/
// -> strg+f wmma

template<typename T> __global__ void matmuladd_simple(T const* const a, T const* const b, T* const c, 
    const int N, const int K, const int M) {
    for (int row = threadIdx.y + blockIdx.y * blockDim.y;
        row < N;
        row += blockDim.y * gridDim.y) {

        for (int col = threadIdx.x + blockIdx.x * blockDim.x;
            col < M;
            col += blockDim.x * gridDim.x) {
            T result = 0;

            for (int k = 0; k < K; k++) {
                result += a[row * N + k] * b[k * M + col];
            }
            c[row * M + col] += result;
        }
    }
}

using namespace nvcuda::wmma;
__global__ void wmma_ker(half* a, half* b, half* c) {
    // declare the fragments
    fragment<matrix_a, 16, 16, 16, half, col_major> a_frag;
    fragment<matrix_b, 16, 16, 16, half, row_major> b_frag;
    fragment<accumulator, 16, 16, 16, half> c_frag;
    // initialize the output to zero
    fill_fragment(c_frag, 0.0f);
    // load the inputs
    load_matrix_sync(a_frag, a, 16);
    load_matrix_sync(b_frag, b, 16);
    // perform the matrix multiplication
    mma_sync(c_frag, a_frag, b_frag, c_frag);
    // store the output
    store_matrix_sync(c, c_frag, 16, mem_row_major);
}

void runMatmulSimple(half* h_a, half* h_b, half* h_c, half* result) {
    half h_d[N * M];
    half* d_a, * d_b, * d_c;
    CHECK_CUDA(hipMalloc(&d_a, N * K * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_b, K * M * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_c, N * M * sizeof(half)));

    CHECK_CUDA(hipMemcpy(d_a, h_a, N * K * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, h_b, K * M * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_c, h_c, N * M * sizeof(half), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(128);
    dim3 blocksPerGrid(16, 1);

    matmuladd_simple<half><<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N, K, M);

    CHECK_CUDA(hipMemcpy(result, d_c, N * M * sizeof(half), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipDeviceSynchronize());
}

void runWMMA(half* h_a, half* h_b, half* h_c, half* result) {
    half h_d[N * M];
    half* d_a, * d_b, * d_c;
    CHECK_CUDA(hipMalloc(&d_a, N * K * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_b, K * M * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_c, N * M * sizeof(half)));

    CHECK_CUDA(hipMemcpy(d_a, h_a, N * K * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, h_b, K * M * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_c, h_c, N * M * sizeof(half), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(128);
    dim3 blocksPerGrid(16, 1);

    wmma_ker<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);

    CHECK_CUDA(hipMemcpy(result, d_c, N * M * sizeof(half), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipDeviceSynchronize());
}

int main()
{
    const int dev = 0;
    std::cout << getCUDADeviceInformations(dev).str() << "\n\n";

    half h_a[N * K]; // N hoch, K breit
    half h_b[K * M]; // K hoch, M breit
    half h_c[N * M]; // N hoch, M breit
    std::srand(1337);

    for (size_t i = 0; i < N * K; i++) {
        h_a[i] = std::rand() / ((RAND_MAX + 1u) / 6);  // Note: 1+rand()%6 is biased
    }
    for (size_t i = 0; i < K * M; i++) {
        h_b[i] = std::rand() / ((RAND_MAX + 1u) / 6);  // Note: 1+rand()%6 is biased
    }
    for (size_t i = 0; i < N * M; i++) {
        h_c[i] = std::rand() / ((RAND_MAX + 1u) / 6);  // Note: 1+rand()%6 is biased
    }

    printMat(h_a, N, K, "h_a");
    printf("\r\n");
    printMat(h_b, K, M, "h_b");
    printf("\r\n");
    printMat(h_c, N, M, "h_c");
    printf("\r\n");
    half simpleResult[N*M];
    runMatmulSimple(h_a, h_b, h_c, simpleResult);
    half wmmaResult[N*M];
    runWMMA(h_a, h_b, h_c, wmmaResult);
    printMat(simpleResult, N, M, "simpleResult");
    printf("\r\n");
    printMat(wmmaResult, N, M, "wmmaResult");
    printf("\r\n");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
